#include "hip/hip_runtime.h"
// Experimental test input for Accelerator directives
// Liao 1/15/2013
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
/* change this to do saxpy or daxpy : single precision or double precision*/
#define REAL double
#define VEC_LEN 1024000 //use a fixed number for now
/* zero out the entire vector */
#include "libxomp.h" 
#include "xomp_cuda_lib_inlined.cu" 

void zero(double *A,int n)
{
  int i;
  for (i = 0; i < n; i++) {
    A[i] = 0.0;
  }
}
/* initialize a vector with random floating point numbers */

void init(double *A,int n)
{
  int i;
  for (i = 0; i < n; i++) {
    A[i] = ((double )(drand48()));
  }
}

__global__ void OUT__1__8164__(int n,double a,double *_dev_x,double *_dev_y)
{
  int _p_i;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init(0,n - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,n - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_i = _dev_lower; _p_i <= _dev_upper; _p_i += 1) {
      _dev_y[_p_i] += a * _dev_x[_p_i];
    }
}

void axpy_ompacc(double *x,double *y,int n,double a)
{
  int i;
/* this one defines both the target device name and data environment to map to,
   I think here we need mechanism to tell the compiler the device type (could be multiple) so that compiler can generate the codes of different versions; 
   we also need to let the runtime know what the target device is so the runtime will chose the right function to call if the code are generated 
   #pragma omp target device (gpu0) map(x, y) 
*/
{
    double *_dev_x;
    int _dev_x_size = sizeof(double ) * (n - 0);
    _dev_x = ((double *)(xomp_deviceMalloc(_dev_x_size)));
    xomp_memcpyHostToDevice(((void *)_dev_x),((const void *)x),_dev_x_size);
    double *_dev_y;
    int _dev_y_size = sizeof(double ) * (n - 0);
    _dev_y = ((double *)(xomp_deviceMalloc(_dev_y_size)));
    xomp_memcpyHostToDevice(((void *)_dev_y),((const void *)y),_dev_y_size);
/* Launch CUDA kernel ... */
    int _threads_per_block_ = xomp_get_maxThreadsPerBlock();
    int _num_blocks_ = xomp_get_max1DBlock(n - 1 - 0 + 1);
    OUT__1__8164__<<<_num_blocks_,_threads_per_block_>>>(n,a,_dev_x,_dev_y);
    xomp_freeDevice(_dev_x);
    xomp_memcpyDeviceToHost(((void *)y),((const void *)_dev_y),_dev_y_size);
    xomp_freeDevice(_dev_y);
  }
}

int main(int argc,char *argv[])
{
  int n;
  double *y_ompacc;
  double *x;
  double a = 123.456;
  n = 1024000;
  y_ompacc = ((double *)(malloc(n * sizeof(double ))));
  x = ((double *)(malloc(n * sizeof(double ))));
  srand48((1 << 12));
  init(x,n);
  init(y_ompacc,n);
/* openmp acc version */
  axpy_ompacc(x,y_ompacc,n,a);
  free(y_ompacc);
  free(x);
  return 0;
}
