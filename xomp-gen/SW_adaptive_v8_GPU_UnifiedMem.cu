#include "hip/hip_runtime.h"
/*********************************************************************************
 * Smith–Waterman algorithm
 * Purpose:     Local alignment of nucleotide or protein sequences
 * Authors:     Daniel Holanda, Hanoch Griner, Taynara Pinheiro
 * Compilation: nvcc -std=c++11 -O3 -DNDEBUG=1 cuda_unified_smithW.cu -o cuda_um_smithW
 *              nvcc -std=c++11 -O0 -DDEBUG -g -G cuda_unified_smithW.cu -o dbg_cuda_smithW
 * Execution:   ./cuda_smithW <number_of_col> <number_of_rows>
 *********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//~ #include <time.h>
//~ #include <omp.h>

#include <cassert>
#include <chrono>
#include <iostream>

#include "parameters.h"

/*--------------------------------------------------------------------
 * Text Tweaks
 */
#define RESET   "\033[0m"
#define BOLDRED "\033[1m\033[31m"      /* Bold Red */
/* End of text tweaks */

/*--------------------------------------------------------------------
 * Constants
 */
#define PATH -1
#define NONE 0
#define UP 1
#define LEFT 2
#define DIAGONAL 3
/* End of constants */

/*--------------------------------------------------------------------
* Helpers
*/
#define min(x, y) (((x) < (y)) ? (x) : (y))
#define max(a,b) ((a) > (b) ? a : b)

// #define DEBUG
/* End of Helpers */

// my types
// \note changed type to unsigned to make it collaborate with CUDA atomicCAS
// \todo maybe rename it to index_t and change all long longs to index_t
typedef unsigned long long maxpos_t;


/*--------------------------------------------------------------------
 * Functions Prototypes
 */
void backtrack(int* P, maxpos_t maxPos);
void printMatrix(int* matrix);
void printPredecessorMatrix(int* matrix);
void generate(void);
long long int nElement(long long int i);

// \pp modified to pass i (a induction variable) by value
void calcFirstDiagElement(long long int i, long long int *si, long long int *sj);

/* End of prototypes */

/*--------------------------------------------------------------------
 * Global Variables
 */
// Defines size of strings to be compared
long long int m = 8; // Columns - Size of string a
long long int n = 9; // Rows    - Size of string b

// Defines scores
static const int       MATCH_SCORE     =  3; //  5 in omp_smithW_orig
static const int       MISSMATCH_SCORE = -3; // -3
static const int       GAP_SCORE       = -2; // -4

// GPU THREADS PER BLOCK
static const long long THREADS_PER_BLOCK = 1024;

// Strings over the Alphabet Sigma
char *a, *b;

/* End of global variables */


/*--------------------------------------------------------------------
 * Function:    matchMissmatchScore
 * Purpose:     Similarity function on the alphabet for match/missmatch
 */
__device__
int matchMissmatchScore_cuda(long long i, long long j, const char* seqa, const char* seqb)
{
    if (seqa[j - 1] == seqb[i - 1])
        return MATCH_SCORE;

    return MISSMATCH_SCORE;
}  /* End of matchMissmatchScore_cuda */

/*--------------------------------------------------------------------
 * Function:    matchMissmatchScore
 * Purpose:     Similarity function on the alphabet for match/missmatch
 */
int matchMissmatchScore(long long int i, long long int j) {
    if (a[j - 1] == b[i - 1])
        return MATCH_SCORE;
    else
        return MISSMATCH_SCORE;
}  /* End of matchMissmatchScore */

/*--------------------------------------------------------------------
 * Function:    SimilarityScore
 * Purpose:     Calculate  the maximum Similarity-Score H(i,j)
 */
__global__
void similarityScore_kernel( long long si,
                             long long sj,
                             long long j_upper_bound,
                             int* H,
                             int* P,
                             maxpos_t* maxPos,
                             const char* seqa,
                             const char* seqb,
                             long long cols
)
{
    // compute the second loop index j
    const long long loopj = blockIdx.x * blockDim.x + threadIdx.x;

    if (loopj >= j_upper_bound) return;

    // compute original i and j
    long long int i = si - loopj;
    long long int j = sj + loopj;

    // bounds test for matchMissmatchScore_cuda
    assert(i > 0); // was: assert(i > 0 && i <= n); -- n currently not passed in
    assert(j > 0 && j <= cols);

    // Stores index of element
    maxpos_t index = cols * i + j;

    assert(index >= cols);
    // Get element above
    int up = H[index - cols] + GAP_SCORE;

    assert(index > 0);
    // Get element on the left
    int left = H[index - 1] + GAP_SCORE;

    assert(index > cols);
    // Get element on the diagonal
    int diag = H[index - cols - 1] + matchMissmatchScore_cuda(i, j, seqa, seqb);

    // Calculates the maximum
    int max  = NONE;
    int pred = NONE;
    /* === Matrix ===
     *      a[0] ... a[n]
     * b[0]
     * ...
     * b[n]
     *
     * generate 'a' from 'b', if '←' insert e '↑' remove
     * a=GAATTCA
     * b=GACTT-A
     *
     * generate 'b' from 'a', if '←' insert e '↑' remove
     * b=GACTT-A
     * a=GAATTCA
    */

    // same letter ↖
    if (diag > max) {
        max = diag;
        pred = DIAGONAL;
    }

    // remove letter ↑
    if (up > max) {
        max = up;
        pred = UP;
    }

    //insert letter ←
    if (left > max) {
        max = left;
        pred = LEFT;
    }

    //Inserts the value in the similarity and predecessor matrixes
    H[index] = max;
    P[index] = pred;

    // Updates maximum score to be used as seed on backtrack
    {
        // \note \pp
        //   locks seem to be a NOGO in CUDA warps,
        //   thus the update to set the maximum is made nonblocking.
        maxpos_t current = *maxPos;
        maxpos_t assumed = current+1;

        while (assumed != current && max > H[current])
        {
            assumed = current;

            // \note consider atomicCAS_system for multi GPU systems
            current = atomicCAS(maxPos, assumed, index);
        }
    }
}  /* End of similarityScore_kernel */


void check_cuda_success(bool cond)
{
    if (cond) return;

    std::cerr << "CUDA error" << std::endl;
    exit(0);
}

/// malloc replacement
template<class T>
static
T* unified_alloc(size_t numelems)
{
    void*       ptr /* = NULL*/;
    hipError_t err = hipMallocManaged(&ptr, numelems * sizeof(T), hipMemAttachGlobal);

    check_cuda_success(err == hipSuccess);
    return reinterpret_cast<T*>(ptr);
}

/// calloc replacement
// \note depending on the OS, the memset may be superfluous.
template<class T>
static
T* unified_alloc_zero(size_t numelems)
{
    T*          ptr = unified_alloc<T>(numelems);
    hipError_t err = hipMemset(ptr, 0, numelems*sizeof(T));

    check_cuda_success(err == hipSuccess);
    return ptr;
}

static
void unified_free(void* ptr)
{
    hipError_t err = hipFree(ptr);

    check_cuda_success(err == hipSuccess);
}

// Start position for backtrack
// \note
//   1) moved out from main function so it can be set in managed space
//   2) made unsigned to fit with CUDA atomicCAS prototype
static __managed__
        maxpos_t maxPos = 0;


void similarityScore_sequential(long long int i, long long int j, int* H, int* P, maxpost_t* maxPos) {

    int up, left, diag;

    //Stores index of element
    long long int index = m * i + j;

    //Get element above
    up = H[index - m] + GAP_SCORE;

    //Get element on the left
    left = H[index - 1] + GAP_SCORE;

    //Get element on the diagonal
    diag = H[index - m - 1] + matchMissmatchScore(i, j);

    //Calculates the maximum
    int max = NONE;
    int pred = NONE;
    /* === Matrix ===
     *      a[0] ... a[n]
     * b[0]
     * ...
     * b[n]
     *
     * generate 'a' from 'b', if '←' insert e '↑' remove
     * a=GAATTCA
     * b=GACTT-A
     *
     * generate 'b' from 'a', if '←' insert e '↑' remove
     * b=GACTT-A
     * a=GAATTCA
    */

    if (diag > max) { //same letter ↖
        max = diag;
        pred = DIAGONAL;
    }

    if (up > max) { //remove letter ↑
        max = up;
        pred = UP;
    }

    if (left > max) { //insert letter ←
        max = left;
        pred = LEFT;
    }
    //Inserts the value in the similarity and predecessor matrixes
    H[index] = max;
    P[index] = pred;

    //Updates maximum score to be used as seed on backtrack
    if (max > H[*maxPos]) {
        *maxPos = index;
    }
}

void similarityScore_ompparallel(long long int i, long long int j, int* H, int* P, maxpos_t * maxPos) {

    int up, left, diag;

    //Stores index of element
    long long int index = m * i + j;

    //Get element above
    up = H[index - m] + GAP_SCORE;

    //Get element on the left
    left = H[index - 1] + GAP_SCORE;

    //Get element on the diagonal
    int t_mms;

    if (a[j - 1] == b[i - 1])
        t_mms = MATCH_SCORE;
    else
        t_mms = MISSMATCH_SCORE;

    diag = H[index - m - 1] + t_mms; // matchMissmatchScore(i, j);

// degug here
// return;
    //Calculates the maximum
    int max = NONE;
    int pred = NONE;
    /* === Matrix ===
     *      a[0] ... a[n]
     * b[0]
     * ...
     * b[n]
     *
     * generate 'a' from 'b', if '←' insert e '↑' remove
     * a=GAATTCA
     * b=GACTT-A
     *
     * generate 'b' from 'a', if '←' insert e '↑' remove
     * b=GACTT-A
     * a=GAATTCA
    */
    if (diag > max) { //same letter ↖
        max = diag;
        pred = DIAGONAL;
    }

    if (up > max) { //remove letter ↑
        max = up;
        pred = UP;
    }

    if (left > max) { //insert letter ←
        max = left;
        pred = LEFT;
    }
    //Inserts the value in the similarity and predecessor matrixes
    H[index] = max;
    P[index] = pred;

    //Updates maximum score to be used as seed on backtrack
    if (max > H[*maxPos]) {
#pragma omp critical
        *maxPos = index;
    }
}

//int MEDIUM=1;
int MEDIUM=1200;
//int LARGE=2048; // max 46340 for GPU of 16GB Device memory
int LARGE=8000; // max 46340 for GPU of 16GB Device memory

/*--------------------------------------------------------------------
 * Function:    main
 */
int main(int argc, char* argv[])
{
    typedef std::chrono::time_point<std::chrono::system_clock> time_point;

    bool useBuiltInData = true;
    if (argc==3)
    {
        m = strtoll(argv[1], NULL, 10);
        n = strtoll(argv[2], NULL, 10);
        useBuiltInData = false;
    } else if (argc == 4)
    {
        m = strtoll(argv[1], NULL, 10);
        n = strtoll(argv[2], NULL, 10);
        LARGE = atoi(argv[3]);
        useBuiltInData = false;
    } else if (argc == 5)
    {
        m = strtoll(argv[1], NULL, 10);
        n = strtoll(argv[2], NULL, 10);
        MEDIUM = strtoll(argv[3], NULL, 10);
        LARGE = strtoll(argv[4], NULL, 10);
        useBuiltInData = false;
    }

//#ifdef DEBUG
    if (useBuiltInData)
        printf ("Using built-in data for testing ..\n");

    printf("Problem size: Matrix[%lld][%lld], FACTOR=%d CUTOFF=%d\n", n, m, FACTOR, CUTOFF);

    // Allocates a and b
    //~ a = malloc(m * sizeof(char));
    //~ b = malloc(n * sizeof(char));
    a = unified_alloc<char>(m+1);
    b = unified_alloc<char>(n+1);

    std::cerr << "a,b allocated: " << m << "/" << n << std::endl;

    // Because now we have zeros
    m++; // \note \pp really needed???
    n++; // \note \pp really needed???

    if (useBuiltInData)
    {
        //Uncomment this to test the sequence available at
        //http://vlab.amrita.edu/?sub=3&brch=274&sim=1433&cnt=1
        // assert(m=11 && n=7);
        // a[0] =   'C';
        // a[1] =   'G';
        // a[2] =   'T';
        // a[3] =   'G';
        // a[4] =   'A';
        // a[5] =   'A';
        // a[6] =   'T';
        // a[7] =   'T';
        // a[8] =   'C';
        // a[9] =   'A';
        // a[10] =  'T';

        // b[0] =   'G';
        // b[1] =   'A';
        // b[2] =   'C';
        // b[3] =   'T';
        // b[4] =   'T';
        // b[5] =   'A';
        // b[6] =   'C';
        // https://en.wikipedia.org/wiki/Smith%E2%80%93Waterman_algorithm#Example
        // Using the wiki example to verify the results
        assert(m>=8 && n>=9);

        b[0] =   'G';
        b[1] =   'G';
        b[2] =   'T';
        b[3] =   'T';
        b[4] =   'G';
        b[5] =   'A';
        b[6] =   'C';
        b[7] =   'T';
        b[8] =   'A';

        a[0] =   'T';
        a[1] =   'G';
        a[2] =   'T';
        a[3] =   'T';
        a[4] =   'A';
        a[5] =   'C';
        a[6] =   'G';
        a[7] =   'G';
    }
    else
    {
        //Gen random arrays a and b
        generate();
    }

    time_point     starttime = std::chrono::system_clock::now();

    // Allocates similarity matrix H
    //~ int* H = calloc(m * n, sizeof(int));
    int* H = unified_alloc_zero<int>(m * n);

    //Allocates predecessor matrix P
    //~ int* P = calloc(m * n, sizeof(int));
    int* P = unified_alloc_zero<int>(m * n);

    // Because now we have zeros ((m-1) + (n-1) - 1)
    long long int nDiag = m + n - 3;

    for (int i = 1; i <= nDiag; ++i)
    {
        long long nEle = nElement(i);
        long long si /* uninitialized */;
        long long sj /* uninitialized */;

        calcFirstDiagElement(i, &si, &sj);

        if (nEle< MEDIUM)
        {
            int j;
            for (j = 0; j < nEle; ++j)
            {  // going upwards : anti-diagnol direction
                long long int ai = si - j ; // going up vertically
                long long int aj = sj + j;  //  going right in horizontal
                similarityScore_sequential(ai, aj, H, P, &maxPos); // a specialized version without a critical section used inside
            }
        }
        else if (nEle<LARGE) // omp cpu version: medium to large: medium data set
        {
#pragma omp for private(j)
            int j;
            for (j = 0; j < nEle; ++j)
            {  // going upwards : anti-diagnol direction
                long long int ai = si - j ; // going up vertically
                long long int aj = sj + j;  //  going right in horizontal
                similarityScore_ompparallel(ai, aj, H, P, &maxPos); // a critical section is used inside
            }
        } else {
            // CUDA, here we go

            // \note
            //   * MAKE SURE THAT a,b,H,P are ACCESSIBLE from GPU.
            //     This prototype allocates a,b,H,P in unified memory space, thus
            //     we just copy the pointers. If the allocation policy changes,
            //     memory referenced by a,b,H,P has to be transferred to the GPU,
            //     and memory referenced by H and P has to be transferred back.
            //   * a and b do not change, thus they only need to be transferred
            //     initially.
            //   * transfers of H and P could probably be optimized to only
            //     include data along the wavefront.
            // \todo
            //   * study amount of data transfer for H and P
            //~ const long long ITER_SPACE = ceil(nEle/THREADS_PER_BLOCK);
            const long long ITER_SPACE = (nEle+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;

            // data transfers :)
            const char* gpuA = a; // only once
            const char* gpuB = b; // only once
            int*        gpuH = H; // only if previous computation was not on GPU
            int*        gpuP = P; // only if previous computation was not on GPU

            // comp. of ai and aj moved into CUDA kernel
            similarityScore_kernel
                    <<<ITER_SPACE, THREADS_PER_BLOCK>>>
                                   (si, sj, nEle, gpuH, gpuP, &maxPos, gpuA, gpuB, m);

            // \todo sync needed?
            //   - not needed when control is not returned to host
            //   - may not be needed at all depending on device capability

            // data transfers :)
            H = gpuH;
            P = gpuP;
        }
    }

    hipDeviceSynchronize();

    time_point     endtime = std::chrono::system_clock::now();

#ifdef DEBUG
    printf("\nSimilarity Matrix:\n");
  printMatrix(H);

  printf("\nPredecessor Matrix:\n");
  printPredecessorMatrix(P);
#endif

    if (useBuiltInData)
    {
        printf ("Verifying results using the builtinIn data: %s\n", (H[n*m-1]==7)?"true":"false");
        assert (H[n*m-1]==7);
    }

    backtrack(P, maxPos);

    int elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(endtime-starttime).count();

    printf("\nElapsed time: %d ms\n\n", elapsed);

    // Frees similarity matrixes
    unified_free(H);
    unified_free(P);

    //Frees input arrays
    unified_free(a);
    unified_free(b);

    return 0;
}  /* End of main */

/*--------------------------------------------------------------------
 * Function:    nElement
 * Purpose:     Calculate the number of i-diagonal elements
 */
long long int nElement(long long int i) {
    if (i < m && i < n) {
        // Number of elements in the diagonal is increasing
        return i;
    }
    else if (i < max(m, n)) {
        //Number of elements in the diagonal is stable
        long int min_mn = min(m, n);
        return min_mn - 1;
    }
    else {
        //Number of elements in the diagonal is decreasing
        long int min_mn = min(m, n);
        return 2 * min_mn - i + abs(m - n) - 2;
    }
}

/*--------------------------------------------------------------------
 * Function:    calcElement
 * Purpose:     Calculate the position of (si, sj)-element
 */
void calcFirstDiagElement(long long int i, long long int *si, long long int *sj) {
    // Calculate the first element of diagonal
    if (i < n) {
        *si = i;
        *sj = 1;
    } else {
        *si = n - 1;
        *sj = i - n + 2;
    }
}



/*--------------------------------------------------------------------
 * Function:    backtrack
 * Purpose:     Modify matrix to print, path change from value to PATH
 */
void backtrack(int* P, maxpos_t maxPos) {
    //hold maxPos value
    long long int predPos = 0;

    std::cerr << "maxpos = " << maxPos << std::endl;

    //backtrack from maxPos to startPos = 0
    do {
        std::cerr << "P[" << maxPos << "] = "
                  << std::flush
                  << P[maxPos]
                  << std::endl;

        switch (P[maxPos])
        {
            case DIAGONAL:
                predPos = maxPos - m - 1;
                break;

            case UP:
                predPos = maxPos - m;
                break;

            case LEFT:
                predPos = maxPos - 1;
                break;

            default:
                assert(false);
        }

        P[maxPos] *= PATH;
        maxPos = predPos;
    } while (P[maxPos] != NONE);
}  /* End of backtrack */

/*--------------------------------------------------------------------
 * Function:    printMatrix
 * Purpose:     Print Matrix
 */
void printMatrix(int* matrix) {
    long long int i, j;
    printf("-\t-\t");
    for (j = 0; j < m-1; j++) {
        printf("%c\t", a[j]);
    }
    printf("\n-\t");
    for (i = 0; i < n; i++) { //Lines
        for (j = 0; j < m; j++) {
            if (j==0 && i>0) printf("%c\t", b[i-1]);
            printf("%d\t", matrix[m * i + j]);
        }
        printf("\n");
    }

}  /* End of printMatrix */

/*--------------------------------------------------------------------
 * Function:    printPredecessorMatrix
 * Purpose:     Print predecessor matrix
 */
void printPredecessorMatrix(int* matrix) {
    long long int i, j, index;
    printf("    ");
    for (j = 0; j < m-1; j++) {
        printf("%c ", a[j]);
    }
    printf("\n  ");
    for (i = 0; i < n; i++) { //Lines
        for (j = 0; j < m; j++) {
            if (j==0 && i>0) printf("%c ", b[i-1]);
            index = m * i + j;
            if (matrix[index] < 0) {
                printf(BOLDRED);
                if (matrix[index] == -UP)
                    printf("↑ ");
                else if (matrix[index] == -LEFT)
                    printf("← ");
                else if (matrix[index] == -DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
                printf(RESET);
            } else {
                if (matrix[index] == UP)
                    printf("↑ ");
                else if (matrix[index] == LEFT)
                    printf("← ");
                else if (matrix[index] == DIAGONAL)
                    printf("↖ ");
                else
                    printf("- ");
            }
        }
        printf("\n");
    }

}  /* End of printPredecessorMatrix */

/*--------------------------------------------------------------------
 * Function:    generate
 * Purpose:     Generate arrays a and b
 */
void generate() {
    //Random seed
    srand(time(NULL));

    //Generates the values of a
    long long int i;
    for (i = 0; i < m; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            a[i] = 'A';
        else if (aux == 2)
            a[i] = 'C';
        else if (aux == 3)
            a[i] = 'G';
        else
            a[i] = 'T';
    }

    //Generates the values of b
    for (i = 0; i < n; i++) {
        int aux = rand() % 4;
        if (aux == 0)
            b[i] = 'A';
        else if (aux == 2)
            b[i] = 'C';
        else if (aux == 3)
            b[i] = 'G';
        else
            b[i] = 'T';
    }
} /* End of generate */


/*--------------------------------------------------------------------
 * External References:
 * http://vlab.amrita.edu/?sub=3&brch=274&sim=1433&cnt=1
 * http://pt.slideshare.net/avrilcoghlan/the-smith-waterman-algorithm
 * http://baba.sourceforge.net/
 */
