#include "hip/hip_runtime.h"
#define NONE 0
#define UP 1
#define LEFT 2
#define DIAGONAL 3
#include "libxomp.h" 
#include "xomp_cuda_lib_inlined.cu" 

#ifdef __cplusplus
extern "C" {
#endif

__global__ void OUT__1__4550__(long long nEle,long long m,long long n,int gapScore,int matchScore,int missmatchScore,long long si,long long sj,char *_dev_a,char *_dev_b,int *_dev_H,int *_dev_P,long long *_dev_maxPos_ptr,int diagonalIndex,int GPUDataOffset)
{
  long long _p_j;
  int _dev_lower;
  int _dev_upper;
  int _dev_loop_chunk_size;
  int _dev_loop_sched_index;
  int _dev_loop_stride;
  int _dev_thread_num = getCUDABlockThreadCount(1);
  int _dev_thread_id = getLoopIndexFromCUDAVariables(1);
  XOMP_static_sched_init((long long )0,nEle - 1,1,1,_dev_thread_num,_dev_thread_id,&_dev_loop_chunk_size,&_dev_loop_sched_index,&_dev_loop_stride);
  while(XOMP_static_sched_next(&_dev_loop_sched_index,nEle - 1,1,_dev_loop_stride,_dev_loop_chunk_size,_dev_thread_num,_dev_thread_id,&_dev_lower,&_dev_upper))
    for (_p_j = _dev_lower; _p_j <= _dev_upper; _p_j += 1) 
// going upwards : anti-diagnol direction
{
// going up vertically
      long long ai = si - _p_j;
//  going right in horizontal
      long long aj = sj + _p_j;
///------------inlined ------------------------------------------
//            similarityScore(ai, aj, H, P, &maxPos); // a critical section is used inside
{
        int up;
        int left;
        int diag;
//Stores index of element
        //long long index = m * ai + aj;
        long long len = min(m,n);
        long long index2 = diagonalIndex*len + GPUDataOffset + _p_j;
//Get element above
        up = _dev_H[index2 - len + 1 - GPUDataOffset] + gapScore;
//Get element on the left
        //left = _dev_H[index2 - len - ((long long )1) - 0] + gapScore;
        left = _dev_H[index2 - len - ((long long )GPUDataOffset) - 0] + gapScore;
//Get element on the diagonal
        int t_mms;
        if (((int )_dev_a[aj - ((long long )1) - 0]) == ((int )_dev_b[ai - ((long long )1) - 0])) 
          t_mms = matchScore;
         else 
          t_mms = missmatchScore;
// matchMissmatchScore(i, j);
        //diag = _dev_H[index - m - ((long long )1) - 0] + t_mms;
        long long temp = index2 - len*2 - ((long long )GPUDataOffset) + ((long long )(1-GPUDataOffset)) - 0;
        //diag = _dev_H[index2 - len*2 - ((long long )GPUDataOffset) + ((long long )(1-GPUDataOffset)) - 0] + t_mms;
        diag = _dev_H[temp] + t_mms;
// degug here
// return;
//Calculates the maximum
        int max = 0;
        int pred = 0;
//same letter ↖
        if (diag > max) {
          max = diag;
          pred = 3;
        }
//remove letter ↑
        if (up > max) {
          max = up;
          pred = 1;
        }
//insert letter ←
        if (left > max) {
          max = left;
          pred = 2;
        }
//Inserts the value in the similarity and predecessor matrixes
        _dev_H[index2 - 0] = max;
        _dev_P[index2 - 0] = pred;
//Updates maximum score to be used as seed on backtrack
  /***** we use cuda atomicCAS to do critical ******
        if (max > _dev_H[_dev_maxPos_ptr[0] - 0]) {
        //#pragma omp critical
          _dev_maxPos_ptr[0 - 0] = index;
        }
        ******/
    {   
    // \note \pp
    //   locks seem to be a NOGO in CUDA warps,
    //   thus the update to set the maximum is made nonblocking.
    unsigned long long int current = _dev_maxPos_ptr[0];
    unsigned long long int assumed = current+1;
#if 0
    while (assumed != current && max > _dev_H[current])
    { 
        assumed = current;

        // \note consider atomicCAS_system for multi GPU systems
        current = atomicCAS((unsigned long long int*)_dev_maxPos_ptr, (unsigned long long int)assumed, (unsigned long long int)index);
    }
#endif 
    } 
      }
// ---------------------------------------------------------------
    }
}

void calculate(char *a,char *b,long long nEle,long long m,long long n,int gapScore,int matchScore,int missmatchScore,long long si,long long sj,int *H,int *P,long long *maxPos_ptr,long long j,int asz,int diagonalIndex,int GPUDataOffset)
{
{
    xomp_deviceDataEnvironmentEnter(0);
    char *_dev_a;
    int _dev_a_size[1] = {m};
    int _dev_a_offset[1] = {0};
    int _dev_a_Dim[1] = {m};
    _dev_a = ((char *)(xomp_deviceDataEnvironmentPrepareVariable(0,(void *)a,1,sizeof(char ),_dev_a_size,_dev_a_offset,_dev_a_Dim,1,0)));
    char *_dev_b;
    int _dev_b_size[1] = {n};
    int _dev_b_offset[1] = {0};
    int _dev_b_Dim[1] = {n};
    _dev_b = ((char *)(xomp_deviceDataEnvironmentPrepareVariable(0,(void *)b,1,sizeof(char ),_dev_b_size,_dev_b_offset,_dev_b_Dim,1,0)));
    int *_dev_H;
    int _dev_H_size[1] = {asz};
    int _dev_H_offset[1] = {0};
    int _dev_H_Dim[1] = {asz};
    _dev_H = ((int *)(xomp_deviceDataEnvironmentPrepareVariable(0,(void *)H,1,sizeof(int ),_dev_H_size,_dev_H_offset,_dev_H_Dim,1,1)));
    int *_dev_P;
    int _dev_P_size[1] = {asz};
    int _dev_P_offset[1] = {0};
    int _dev_P_Dim[1] = {asz};
    _dev_P = ((int *)(xomp_deviceDataEnvironmentPrepareVariable(0,(void *)P,1,sizeof(int ),_dev_P_size,_dev_P_offset,_dev_P_Dim,1,1)));
    long long *_dev_maxPos_ptr;
    int _dev_maxPos_ptr_size[1] = {1};
    int _dev_maxPos_ptr_offset[1] = {0};
    int _dev_maxPos_ptr_Dim[1] = {1};
    _dev_maxPos_ptr = ((long long *)(xomp_deviceDataEnvironmentPrepareVariable(0,(void *)maxPos_ptr,1,sizeof(long long ),_dev_maxPos_ptr_size,_dev_maxPos_ptr_offset,_dev_maxPos_ptr_Dim,1,1)));
/* Launch CUDA kernel ... */
    int _threads_per_block_ = xomp_get_maxThreadsPerBlock(0);
    int _num_blocks_ = xomp_get_max1DBlock(0,nEle - 1 - ((long long )0) + 1);
    OUT__1__4550__<<<_num_blocks_,_threads_per_block_>>>(nEle,m,n,gapScore,matchScore,missmatchScore,si,sj,_dev_a,_dev_b,_dev_H,_dev_P,_dev_maxPos_ptr,diagonalIndex,GPUDataOffset);
    xomp_deviceDataEnvironmentExit(0);
  }
}
//      } // for end nDiag
//    } // end omp parallel
//}
#ifdef __cplusplus
}
#endif
